#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

// CUDA kernel for element-wise multiplication
__global__ void elementwise_multiply_kernel(const double *a, const double *b, double *c, size_t total_elements) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total_elements) {
        c[idx] = a[idx] * b[idx];
    }
}

// Host wrapper function for CUDA kernel
extern "C" void elementwise_multiply_gpu(const double *a, const double *b, double *c, size_t total_elements) {
    double *d_a = NULL, *d_b = NULL, *d_c = NULL;
    size_t size = total_elements * sizeof(double);

    // Allocate device memory
    if (hipMalloc((void **)&d_a, size) != hipSuccess) {
        printf("CUDA malloc failed for d_a\n");
        return;
    }
    if (hipMalloc((void **)&d_b, size) != hipSuccess) {
        printf("CUDA malloc failed for d_b\n");
        hipFree(d_a);
        return;
    }
    if (hipMalloc((void **)&d_c, size) != hipSuccess) {
        printf("CUDA malloc failed for d_c\n");
        hipFree(d_a);
        hipFree(d_b);
        return;
    }

    // Copy data from host to device
    if (hipMemcpy(d_a, a, size, hipMemcpyHostToDevice) != hipSuccess) {
        printf("CUDA memcpy failed for d_a\n");
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
        return;
    }
    if (hipMemcpy(d_b, b, size, hipMemcpyHostToDevice) != hipSuccess) {
        printf("CUDA memcpy failed for d_b\n");
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
        return;
    }

    // Configure CUDA kernel launch parameters
    size_t threads_per_block = 256;
    size_t blocks = (total_elements + threads_per_block - 1) / threads_per_block;

    // Launch CUDA kernel
    elementwise_multiply_kernel<<<blocks, threads_per_block>>>(d_a, d_b, d_c, total_elements);

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
        return;
    }

    // Wait for device to finish and check for errors
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA synchronization error: %s\n", hipGetErrorString(err));
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
        return;
    }

    // Copy result back to host
    if (hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost) != hipSuccess) {
        printf("CUDA memcpy failed for d_c\n");
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
        return;
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
